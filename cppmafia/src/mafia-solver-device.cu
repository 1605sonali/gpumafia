#include "hip/hip_runtime.h"
/** @file mafia-solver-device.cpp device-related parts of MAFIA solver
		this file is not compiled if compiling without GPU support
 */

#include "mafia-solver.h"

#ifdef MAFIA_USE_DEVICE

#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>
#include <thrust/reduce.h>

using namespace thrust;

template<class T> void MafiaSolver<T>::touch_dev() {
	char h_arr[1], *d_arr;
	CHECK(hipMalloc((void**)&d_arr, 1));
	CHECK(hipMemcpy(d_arr, h_arr, 1, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(h_arr, d_arr, 1, hipMemcpyDeviceToHost));
	hipFree(d_arr);
}  // touch_dev()

template<class T> void MafiaSolver<T>::copy_ps_to_device() {
	CHECK(hipMalloc((void**)&d_ps, sizeof(*d_ps) * n * d));
	CHECK(hipMemcpy(d_ps, ps, sizeof(*d_ps) * n * d, hipMemcpyHostToDevice));
}

template<class T> void MafiaSolver<T>::compute_limits_dev() {
	//T *d_pmins, *d_pmaxs;
	//size_t limits_sz = sizeof(*d_pmins) * d;
	//CHECK(hipMalloc((void**)&d_pmins, limits_sz));
	//CHECK(hipMalloc((void**)&d_pmaxs, limits_sz));
	device_ptr<T> d_psptr(d_ps);
	for(int idim = 0; idim < d; idim++) {
		pmins[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 std::numeric_limits<T>::infinity(), minimum<T>());
		pmaxs[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 -std::numeric_limits<T>::infinity(), maximum<T>());
	}	
	//CHECK(hipMemcpy(pmins, d_pmins, limits_sz, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(pmaxs, d_pmaxs, limits_sz, hipMemcpyDeviceToHost));
	//hipFree(pmins);
	//hipFree(pmaxs);
}  // compute_limits_dev

/** histogram computation kernel 
		@param psd a single dimension of the points on device
		@param n the number of points on the device
		@param pmin the minimum coordinate of the point along the current dimension
		@param piwidth the inverted width of the bin along the current dimension
		@param histo the histogram along the current dimension
		@param nbins the number of bins along the current dimension
*/
template<class T>
__global__ void histo_kernel
(T* psd, int n, double pmin, double piwidth, int *histo, int nbins) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n)
		return;
	int ibin = (int)floor((psd[i] - pmin) * piwidth);
	ibin = min(max(ibin, 0), nbins - 1);
	atomicAdd(histo + ibin, 1);
}  // histo_kernel
// instantiations
template __global__ void histo_kernel
(float* psd, int n, double pmin, double piwidth, int *histo, int nbins);
template __global__ void histo_kernel
(double* psd, int n, double pmin, double piwidth, int *histo, int nbins);

/** bins in shared memory
		@param nels number of input elements to be processed by each input thread
 */
extern __shared__ int lhisto[];
template<class T>
__global__ void local_histo_kernel
(T* psd, int n, double pmin, double piwidth, int *histo, int nbins, int nels) {
	// zero shared memory bins
	int ii = threadIdx.x;
	int bs = blockDim.x;
	for(int ibin = ii; ibin < nbins; ibin += bs) 
		lhisto[ibin] = 0;
	__syncthreads();
	// compute the histogram in shared memory
	int istart = ii + blockIdx.x * bs * nels;
	int iend = min(ii + (blockIdx.x + 1) * bs * nels, n);
	for(int i = istart; i < iend; i += bs) {
		int ibin = (int)floor((psd[i] - pmin) * piwidth);
		ibin = min(max(ibin, 0), nbins - 1);
		atomicAdd(lhisto + ibin, 1);		
	}
	__syncthreads();
	// accumulate the histogram into the global memory
	for(int ibin = ii; ibin < nbins; ibin += bs)
		atomicAdd(histo + ibin, *(lhisto + ibin));
}  // local_histo_kernel

template<class T> void MafiaSolver<T>::compute_histo_dev(int idim) {
	int nbins = nbinss[idim];
	// on-device data for a histogram
	int *d_histo;
	size_t histo_sz = sizeof(*d_histo) * nbins;
	CHECK(hipMalloc((void**)&d_histo, histo_sz));
	CHECK(hipMemset(d_histo, 0, histo_sz));
	// kernel call
	// TODO: handle more than 256M points
	size_t bs = 256;
	//histo_kernel<<<divup(n, bs), bs>>>
	//	(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]), 
	//	 d_histo, nbins);	
	int nels = 16;
	local_histo_kernel<<<divup(n, bs * nels), bs, nbins * sizeof(int)>>>
		(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]), 
		 d_histo, nbins, nels);	
	CHECK(hipDeviceSynchronize());
	
	// copy the data back
	CHECK(hipMemcpy(histos[idim], d_histo, histo_sz, hipMemcpyDeviceToHost));
	hipFree(d_histo);
}  // compute_histo_dev

/** kernel to compute the bitmaps on device 
		@param bmp bitmap data on device
		@param nwords number of 32-bit words in the bitmap
		@param ps device point data for thebitmap's dimension
		@param n the number of points
		@param pleft the left boundary of the window's range (inclusive)
		@param pright the right boundary of the window's range (non-inclusive)
 */
template<class T> __global__ void bitmap_kernel
(unsigned *bmp, int nwords, T *ps, int n, T pleft, T pright) {
	//int iword = threadIdx.x + blockIdx.x * blockDim.x;
	// TODO: use local memory
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n)
		return;
	int bit = pleft <= ps[i] && ps[i] < pright ? 1 : 0;
	int shift = i % (sizeof(int) * 8);
	atomicOr(bmp + i / (sizeof(int) * 8), bit << shift);
	/* int iword = threadIdx.x + blockIdx.x * blockDim.x;
	if(iword >= nwords)
		return;
	// accumulate the data word over the points
	int word = 0; 
	int istart = iword * sizeof(int) * 8; 
	int iend = min(n, (iword + 1) * (int)sizeof(int) * 8);
	for(int i = istart; i < iend; i++) {
		int bit = pleft <= ps[i] && ps[i] < pright ? 1 : 0;
		word |= bit << (i - istart);
	}  // for(i)
	bmp[iword] = word; */
}  // bitmap_kernel
// instantiations
template __global__ void bitmap_kernel
(unsigned *bmp, int nwords, float *ps, int n, float pleft, float pright);
template __global__ void bitmap_kernel
(unsigned *bmp, int nwords, double *ps, int n, double pleft, double pright);

template<class T> 
void MafiaSolver<T>::compute_bitmap_dev(int idim, int iwin) {
	Window &w = windows[idim][iwin];
	// allocate memory on device
	unsigned *d_bmp;
	int nwords = w.pset->n, bmp_sz = nwords * sizeof(*d_bmp);
	CHECK(hipMalloc((void**)&d_bmp, bmp_sz));
	CHECK(hipMemset(d_bmp, 0, bmp_sz));
	// call the kernel
	size_t bs = 256;
	//bitmap_kernel<<<divup(nwords, bs), bs>>>
	//	(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	bitmap_kernel<<<divup(n, bs), bs>>>
		(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	hipDeviceSynchronize();

	// copy data back
	CHECK(hipMemcpy(w.pset->data, d_bmp, bmp_sz, hipMemcpyDeviceToHost));
	hipFree(d_bmp);
}  // compute_bitmap_dev

template<class T>
void MafiaSolver<T>::free_dev_resources() {
	hipFree(d_ps);
}  // free_dev_resources

// explicit instantiations
template class MafiaSolver<float>;
template class MafiaSolver<double>;
  
#endif
