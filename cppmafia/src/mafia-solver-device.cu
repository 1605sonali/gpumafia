#include "hip/hip_runtime.h"
/** @file mafia-solver-device.cpp device-related parts of MAFIA solver
		this file is not compiled if compiling without GPU support
 */

#include "mafia-solver.h"

#ifdef MAFIA_USE_DEVICE

#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>
#include <thrust/reduce.h>

#define MAX_CDU_DIM 32
#define CDUS_PER_BLOCK 8
#define MAX_GRID_BLOCKS_Y 32768
#define PCOUNT_NWORDS_PTHREAD 32

using namespace thrust;

template<class T> void MafiaSolver<T>::touch_dev() {
	char h_arr[1], *d_arr;
	CHECK(hipMalloc((void**)&d_arr, 1));
	CHECK(hipMemcpy(d_arr, h_arr, 1, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(h_arr, d_arr, 1, hipMemcpyDeviceToHost));
	hipFree(d_arr);
}  // touch_dev()

template<class T> void MafiaSolver<T>::copy_ps_to_device() {
	CHECK(hipMalloc((void**)&d_ps, sizeof(*d_ps) * n * d));
	CHECK(hipMemcpy(d_ps, ps, sizeof(*d_ps) * n * d, hipMemcpyHostToDevice));
}

template<class T> void MafiaSolver<T>::compute_limits_dev() {
	// compute limits using thrust
	device_ptr<T> d_psptr(d_ps);
	for(int idim = 0; idim < d; idim++) {
		pmins[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 std::numeric_limits<T>::infinity(), minimum<T>());
		pmaxs[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 -std::numeric_limits<T>::infinity(), maximum<T>());
	}	
}  // compute_limits_dev

/** histogram computation kernel 
		@param psd a single dimension of the points on device
		@param n the number of points on the device
		@param pmin the minimum coordinate of the point along the current dimension
		@param piwidth the inverted width of the bin along the current dimension
		@param histo the histogram along the current dimension
		@param nbins the number of bins along the current dimension
*/
template<class T, int nels, int blocksize>
__launch_bounds__(blocksize)
__global__ void histo_kernel
(const T* __restrict__ const psd, const int n, const double pmin, const double piwidth, int* __restrict__ const histo, const int nbins)
{
	const int start = blockIdx.x*blocksize*nels + threadIdx.x;
	#pragma unroll
	for(int i = 0; i < nels; ++i)
	{
		int idx = start + i * blocksize;
		if ( idx < n )
		{
			int ibin = (int)floor((psd[idx] - pmin) * piwidth);
			ibin = min(max(ibin, 0), nbins - 1);
			atomicAdd(histo + ibin, 1);
		}
	}
}  // histo_kernel

/** bins in shared memory
		@param nels number of input elements to be processed by each input thread
 */
extern __shared__ int lhisto[];
template<class T>
__global__ void local_histo_kernel
(const T* __restrict__ const psd, const int n, const double pmin, const double piwidth, int* __restrict__ const histo, const int nbins, const int nels) {
	// zero shared memory bins
	int ii = threadIdx.x;
	int bs = blockDim.x;
	for(int ibin = ii; ibin < nbins; ibin += bs) 
		lhisto[ibin] = 0;
	__syncthreads();
	// compute the histogram in shared memory
	int istart = ii + blockIdx.x * bs * nels;
	int iend = min(ii + (blockIdx.x + 1) * bs * nels, n);
	for(int i = istart; i < iend; i += bs) {
		int ibin = (int)floor((psd[i] - pmin) * piwidth);
		ibin = min(max(ibin, 0), nbins - 1);
		atomicAdd(lhisto + ibin, 1);		
	}
	__syncthreads();
	// accumulate the histogram into the global memory
	for(int ibin = ii; ibin < nbins; ibin += bs)
		atomicAdd(histo + ibin, *(lhisto + ibin));
}  // local_histo_kernel

template<class T> void MafiaSolver<T>::compute_histo_dev(int idim) {
	int nbins = nbinss[idim];
	// on-device data for a histogram
	size_t histo_sz = sizeof(*d_histo) * nbins;
	if (d_hist_size < histo_sz) {
		hipFree(d_histo);
		CHECK(hipMalloc((void**)&d_histo, histo_sz));
		d_hist_size = histo_sz;
	}
	CHECK(hipMemset(d_histo, 0, histo_sz));
	// kernel call
	// TODO: handle more than 256M points
	
#ifdef KEPLER_OPTIMIZATIONS
	int nels = 4;
	size_t bs = 256;
	histo_kernel<T,4,256><<<divup(n, bs * nels ), bs>>>
		(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]),
		 d_histo, nbins);
#else //!KEPLER_OPTIMIZATIONS
	int nels = 16;
	size_t bs = 256;
	local_histo_kernel<<<divup(n, bs * nels), bs, nbins * sizeof(int)>>>
		(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]),
		 d_histo, nbins, nels);
#endif
	CHECK(hipGetLastError());
	
	// copy the data back
	CHECK(hipMemcpy(histos[idim], d_histo, histo_sz, hipMemcpyDeviceToHost));
}  // compute_histo_dev

template<class T>
void MafiaSolver<T>::alloc_bitmaps_dev() {
	int nwindows = dense_ws.size();
	CHECK(hipMalloc((void**)&d_bmps, sizeof(*d_bmps) * nwindows * nwords));
	CHECK(hipMemset(d_bmps, 0, sizeof(*d_bmps) * nwindows * nwords));
}  // alloc_bitmaps_dev

/** kernel to compute the bitmaps on device
	@param bmp bitmap data on device
	@param nwords number of 32-bit words in the bitmap
	@param ps device point data for thebitmap's dimension
	@param n the number of points
	@param pleft the left boundary of the window's range (inclusive)
	@param pright the right boundary of the window's range (non-inclusive)
 */
template<class T> __global__ void bitmap_kernel
(unsigned * __restrict__ const bmp, const int nwords, const T* __restrict__ const ps, const int n, const T pleft, const T pright) {
	//int iword = threadIdx.x + blockIdx.x * blockDim.x;
	// TODO: use local memory
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n)
		return;
	const T psv = ps[i];
	const int bit = pleft <= psv && psv < pright ? 1 : 0;
	const int shift = i % (sizeof(int) * 8);
	atomicOr(bmp + i / (sizeof(int) * 8), bit << shift);
	/* int iword = threadIdx.x + blockIdx.x * blockDim.x;
	if(iword >= nwords)
		return;
	// accumulate the data word over the points
	int word = 0;
	int istart = iword * sizeof(int) * 8;
	int iend = min(n, (iword + 1) * (int)sizeof(int) * 8);
	for(int i = istart; i < iend; i++) {
		int bit = pleft <= ps[i] && ps[i] < pright ? 1 : 0;
		word |= bit << (i - istart);
	}  // for(i)
	bmp[iword] = word; */
}  // bitmap_kernel
// instantiations

template<class T>
void MafiaSolver<T>::compute_bitmap_dev(int iwin) {
	Window &w = dense_ws[iwin];
	int idim = w.idim;
	// allocate memory on device
	unsigned *h_bmp = bmps + iwin * nwords, *d_bmp = d_bmps + iwin * nwords;
	// call the kernel
	size_t bs = 256;
	//bitmap_kernel<<<divup(nwords, bs), bs>>>
	//	(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	bitmap_kernel<<<divup(n, bs), bs>>>
		(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	CHECK(hipGetLastError());

	// copy data back
	int bmp_sz = nwords * sizeof(*d_bmp);
	CHECK(hipMemcpy(h_bmp, d_bmp, bmp_sz, hipMemcpyDeviceToHost));
}  // compute_bitmap_dev

__global__ void point_count_kernel
(int* __restrict__ const pcounts, const int* __restrict__ const iwins, const int ncdus, const int ncoords, const unsigned* __restrict__ const bmps, const int nwords,
 const int nwords_pthr) {
       	__shared__ int liwins[CDUS_PER_BLOCK][MAX_CDU_DIM];
	int licdu = threadIdx.y;
	int icdu = licdu + blockIdx.y * blockDim.y;
	int bs = blockDim.x;
	int istart = threadIdx.x + blockIdx.x * bs * nwords_pthr;
	int iend = min(istart + nwords_pthr * bs, nwords);
	// load window bmp starts to local memory
	int licoord = threadIdx.x;
	if(icdu < ncdus && licoord < ncoords) {
		liwins[licdu][licoord] = 
			iwins[icdu * ncoords + licoord] * nwords;
	}
	__syncthreads();
	if(icdu >= ncdus)
		return;
	int pcount = 0;
	for(int iword = istart; iword < iend; iword += bs) {
		unsigned word = ~0u;
		for(int icoord = 0; icoord < ncoords; icoord++)
			word &= bmps[liwins[licdu][icoord] + iword];
		pcount += __popc(word);
	}
	atomicAdd(pcounts + icdu, pcount);
}  // point_count_kernel

template<class T>
void MafiaSolver<T>::count_points_dev() {
	// copy CDU data to device, first aggregate on host; just window indices will do
	int ncdus = cdus.size();
	int ncoords = cur_dim + 1;
	// window numbers, in CDU-major order
	size_t win_sz = sizeof(int) * ncdus * ncoords;
	if ( hd_iwins_sz < win_sz )
	{
		bulk_free(h_iwins);
		hipFree(d_iwins);

		h_iwins = (int*)bulk_alloc(win_sz);
		CHECK(hipMalloc((void**)&d_iwins, win_sz));
		hd_iwins_sz = win_sz;
	}
	//TODO: would it make sense to do this in a pipeline?
	for(int icdu = 0; icdu < ncdus; icdu++) {
		Cdu &cdu = *cdus[icdu];
		for(int icoord = 0; icoord < ncoords; icoord++)
			h_iwins[icdu * ncoords + icoord] = cdu.coords[icoord].win;
	}
	CHECK(hipMemcpy(d_iwins, h_iwins, win_sz, hipMemcpyHostToDevice));

	// run the kernel on device
	size_t pcount_sz = sizeof(*h_pcounts) * ncdus;
	if ( hd_pcounts_sz < pcount_sz )
	{
		hipFree(d_pcounts);
		bulk_free(h_pcounts);
		CHECK(hipMalloc((void**)&d_pcounts, pcount_sz));
		h_pcounts = (int*)bulk_alloc(pcount_sz);
		hd_pcounts_sz = pcount_sz;
	}
	CHECK(hipMemset(d_pcounts, 0, pcount_sz));
	//This value can be lowered to lanuch more threads (less work per thread)
	int nwords_pthr = min(max(nwords / PCOUNT_NWORDS_PTHREAD, 2), 
	  PCOUNT_NWORDS_PTHREAD); // number of words per thread
	dim3 bs(64, CDUS_PER_BLOCK);  // block size
	// iterate over CDU parts
	int ncdus_ppart = MAX_GRID_BLOCKS_Y * bs.y;
	int ncdu_parts = divup(ncdus, ncdus_ppart);
	for(int icdu_part = 0; icdu_part < ncdu_parts; icdu_part++) {
		int cur_ncdus = min(ncdus_ppart, ncdus - icdu_part * ncdus_ppart);
		dim3 grid(divup(nwords, nwords_pthr * bs.x), divup(cur_ncdus, bs.y));
		// TODO: support more than 2**18-4 CDUs
		point_count_kernel<<<grid, bs>>>
			(d_pcounts + icdu_part * ncdus_ppart, d_iwins + icdu_part * ncdus_ppart, cur_ncdus, ncoords,
			 d_bmps, nwords, nwords_pthr);
	}
	//TODO: replace with hipGetLastError: Why does it crash in this case?
	CHECK(hipDeviceSynchronize());

	// copy data back
	//TODO: is it resonalbe to do this piplelined?
	CHECK(hipMemcpy(h_pcounts, d_pcounts, pcount_sz, hipMemcpyDeviceToHost));
	for(int icdu = 0; icdu < ncdus; icdu++)
		cdus[icdu]->npoints = h_pcounts[icdu];
}  // count_points_dev

template<class T>
void MafiaSolver<T>::free_dev_resources() {
	hipFree(d_ps);
	hipFree(d_bmps);
	hipFree(d_histo);
	hipFree(d_iwins);
	hipFree(d_pcounts);
	bulk_free(h_pcounts);
	bulk_free(h_iwins);
}  // free_dev_resources

// explicit instantiations
template class MafiaSolver<float>;
template class MafiaSolver<double>;

#endif
