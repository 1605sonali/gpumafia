#include "hip/hip_runtime.h"
/** @file mafia-solver-device.cpp device-related parts of MAFIA solver
		this file is not compiled if compiling without GPU support
 */

#include "mafia-solver.h"

#ifdef MAFIA_USE_DEVICE

#include <hip/hip_runtime.h>
#include <limits>
#include <stdio.h>
#include <thrust/reduce.h>

using namespace thrust;

template<class T> void MafiaSolver<T>::touch_dev() {
	char h_arr[1], *d_arr;
	CHECK(hipMalloc((void**)&d_arr, 1));
	CHECK(hipMemcpy(d_arr, h_arr, 1, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(h_arr, d_arr, 1, hipMemcpyDeviceToHost));
	hipFree(d_arr);
}  // touch_dev()

template<class T> void MafiaSolver<T>::copy_ps_to_device() {
	CHECK(hipMalloc((void**)&d_ps, sizeof(*d_ps) * n * d));
	CHECK(hipMemcpy(d_ps, ps, sizeof(*d_ps) * n * d, hipMemcpyHostToDevice));
}

template<class T> void MafiaSolver<T>::compute_limits_dev() {
	// compute limits using thrust
	device_ptr<T> d_psptr(d_ps);
	for(int idim = 0; idim < d; idim++) {
		pmins[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 std::numeric_limits<T>::infinity(), minimum<T>());
		pmaxs[idim] = reduce(d_psptr + idim * n, d_psptr + (idim + 1) * n, 
												 -std::numeric_limits<T>::infinity(), maximum<T>());
	}	
}  // compute_limits_dev

/** histogram computation kernel 
		@param psd a single dimension of the points on device
		@param n the number of points on the device
		@param pmin the minimum coordinate of the point along the current dimension
		@param piwidth the inverted width of the bin along the current dimension
		@param histo the histogram along the current dimension
		@param nbins the number of bins along the current dimension
*/
template<class T>
__global__ void histo_kernel
(T* psd, int n, double pmin, double piwidth, int *histo, int nbins) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n)
		return;
	int ibin = (int)floor((psd[i] - pmin) * piwidth);
	ibin = min(max(ibin, 0), nbins - 1);
	atomicAdd(histo + ibin, 1);
}  // histo_kernel
// instantiations
template __global__ void histo_kernel
(float* psd, int n, double pmin, double piwidth, int *histo, int nbins);
template __global__ void histo_kernel
(double* psd, int n, double pmin, double piwidth, int *histo, int nbins);

/** bins in shared memory
		@param nels number of input elements to be processed by each input thread
 */
extern __shared__ int lhisto[];
template<class T>
__global__ void local_histo_kernel
(T* psd, int n, double pmin, double piwidth, int *histo, int nbins, int nels) {
	// zero shared memory bins
	int ii = threadIdx.x;
	int bs = blockDim.x;
	for(int ibin = ii; ibin < nbins; ibin += bs) 
		lhisto[ibin] = 0;
	__syncthreads();
	// compute the histogram in shared memory
	int istart = ii + blockIdx.x * bs * nels;
	int iend = min(ii + (blockIdx.x + 1) * bs * nels, n);
	for(int i = istart; i < iend; i += bs) {
		int ibin = (int)floor((psd[i] - pmin) * piwidth);
		ibin = min(max(ibin, 0), nbins - 1);
		atomicAdd(lhisto + ibin, 1);		
	}
	__syncthreads();
	// accumulate the histogram into the global memory
	for(int ibin = ii; ibin < nbins; ibin += bs)
		atomicAdd(histo + ibin, *(lhisto + ibin));
}  // local_histo_kernel

template<class T> void MafiaSolver<T>::compute_histo_dev(int idim) {
	int nbins = nbinss[idim];
	// on-device data for a histogram
	int *d_histo;
	size_t histo_sz = sizeof(*d_histo) * nbins;
	CHECK(hipMalloc((void**)&d_histo, histo_sz));
	CHECK(hipMemset(d_histo, 0, histo_sz));
	// kernel call
	// TODO: handle more than 256M points
	size_t bs = 256;
	//histo_kernel<<<divup(n, bs), bs>>>
	//	(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]), 
	//	 d_histo, nbins);	
	int nels = 16;
	local_histo_kernel<<<divup(n, bs * nels), bs, nbins * sizeof(int)>>>
		(d_ps + idim * n, n, pmins[idim], nbins / (pmaxs[idim] - pmins[idim]), 
		 d_histo, nbins, nels);	
	CHECK(hipDeviceSynchronize());
	
	// copy the data back
	CHECK(hipMemcpy(histos[idim], d_histo, histo_sz, hipMemcpyDeviceToHost));
	hipFree(d_histo);
}  // compute_histo_dev

template<class T> 
void MafiaSolver<T>::alloc_bitmaps_dev() {
	int nwindows = dense_ws.size();
	CHECK(hipMalloc((void**)&d_bmps, sizeof(*d_bmps) * nwindows * nwords));
	CHECK(hipMemset(d_bmps, 0, sizeof(*d_bmps) * nwindows * nwords));
}  // alloc_bitmaps_dev

/** kernel to compute the bitmaps on device 
		@param bmp bitmap data on device
		@param nwords number of 32-bit words in the bitmap
		@param ps device point data for thebitmap's dimension
		@param n the number of points
		@param pleft the left boundary of the window's range (inclusive)
		@param pright the right boundary of the window's range (non-inclusive)
 */
template<class T> __global__ void bitmap_kernel
(unsigned *bmp, int nwords, T *ps, int n, T pleft, T pright) {
	//int iword = threadIdx.x + blockIdx.x * blockDim.x;
	// TODO: use local memory
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i >= n)
		return;
	int bit = pleft <= ps[i] && ps[i] < pright ? 1 : 0;
	int shift = i % (sizeof(int) * 8);
	atomicOr(bmp + i / (sizeof(int) * 8), bit << shift);
	/* int iword = threadIdx.x + blockIdx.x * blockDim.x;
	if(iword >= nwords)
		return;
	// accumulate the data word over the points
	int word = 0; 
	int istart = iword * sizeof(int) * 8; 
	int iend = min(n, (iword + 1) * (int)sizeof(int) * 8);
	for(int i = istart; i < iend; i++) {
		int bit = pleft <= ps[i] && ps[i] < pright ? 1 : 0;
		word |= bit << (i - istart);
	}  // for(i)
	bmp[iword] = word; */
}  // bitmap_kernel
// instantiations
template __global__ void bitmap_kernel
(unsigned *bmp, int nwords, float *ps, int n, float pleft, float pright);
template __global__ void bitmap_kernel
(unsigned *bmp, int nwords, double *ps, int n, double pleft, double pright);

template<class T> 
void MafiaSolver<T>::compute_bitmap_dev(int iwin) {
	Window &w = dense_ws[iwin];
	int idim = w.idim;
	// allocate memory on device
	unsigned *h_bmp = bmps + iwin * nwords, *d_bmp = d_bmps + iwin * nwords;
	// call the kernel
	size_t bs = 256;
	//bitmap_kernel<<<divup(nwords, bs), bs>>>
	//	(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	bitmap_kernel<<<divup(n, bs), bs>>>
		(d_bmp, nwords, d_ps + n * idim, n, (T)w.pleft, (T)w.pright);
	CHECK(hipDeviceSynchronize());

	// copy data back
	int bmp_sz = nwords * sizeof(*d_bmp);
	CHECK(hipMemcpy(h_bmp, d_bmp, bmp_sz, hipMemcpyDeviceToHost));
}  // compute_bitmap_dev

__global__ void point_count_kernel
(int *pcounts, int *iwins, int ncdus, int ncoords, unsigned *bmps, int nwords,
 int nwords_pthr) {
	int icdu = threadIdx.y + blockIdx.y * blockDim.y;
	if(icdu >= ncdus)
		return;
	int bs = blockDim.x;
	int istart = threadIdx.x + blockIdx.x * bs * nwords_pthr;
	int iend = min(istart + nwords_pthr * bs, nwords);
	int pcount = 0;
	for(int iword = istart; iword < iend; iword += bs) {
		unsigned word = ~0u;
		for(int icoord = 0; icoord < ncoords; icoord++) {
			int iwin = iwins[icdu * ncoords + icoord];
			word &= bmps[iwin * nwords + iword];
		}
		pcount += __popc(word);
	}
	atomicAdd(pcounts + icdu, pcount);
}  // point_count_kernel

template<class T>
void MafiaSolver<T>::count_points_dev() {
	// copy CDU data to device, first aggregate on host; just window indices will do
	int ncdus = cdus.size();
	int ncoords = cur_dim + 1;
	// window numbers, in CDU-major order
	size_t win_sz = sizeof(int) * ncdus * ncoords;
	int *h_iwins = (int*)bulk_alloc(win_sz);
	for(int icdu = 0; icdu < ncdus; icdu++) {
		Cdu &cdu = *cdus[icdu];
		for(int icoord = 0; icoord < ncoords; icoord++)
			h_iwins[icdu * ncoords + icoord] = cdu.coords[icoord].win;
	}
	int *d_iwins; // window numbers, on device
	CHECK(hipMalloc((void**)&d_iwins, win_sz));
	CHECK(hipMemcpy(d_iwins, h_iwins, win_sz, hipMemcpyHostToDevice));
	bulk_free(h_iwins);
	
	// run the kernel on device
	int *h_pcounts, *d_pcounts;  // point counts on host and device
	size_t pcount_sz = sizeof(*h_pcounts) * ncdus;
	CHECK(hipMalloc((void**)&d_pcounts, pcount_sz));
	CHECK(hipMemset(d_pcounts, 0, pcount_sz));
	int nwords_pthr = min(max(nwords / 64, 2), 64); // number of words per thread
	dim3 bs(64, 8);  // block size
	// iterate over CDU parts
	int ncdus_ppart = 32768 * bs.y;
	int ncdu_parts = divup(ncdus, ncdus_ppart);
	for(int icdu_part = 0; icdu_part < ncdu_parts; icdu_part++) {
		int cur_ncdus = min(ncdus_ppart, ncdus - icdu_part * ncdus_ppart);
		dim3 grid(divup(nwords, nwords_pthr * bs.x), divup(cur_ncdus, bs.y));
		// TODO: support more than 2**18-4 CDUs
		point_count_kernel<<<grid, bs>>>
			(d_pcounts + icdu_part * ncdus_ppart, d_iwins + icdu_part * ncdus_ppart, cur_ncdus, ncoords, 
			 d_bmps, nwords, nwords_pthr);
	}
	CHECK(hipDeviceSynchronize());

	// copy data back
	h_pcounts = (int*)bulk_alloc(pcount_sz);
	CHECK(hipMemcpy(h_pcounts, d_pcounts, pcount_sz, hipMemcpyDeviceToHost));
	for(int icdu = 0; icdu < ncdus; icdu++)
		cdus[icdu]->npoints = h_pcounts[icdu];

	// free everything
	hipFree(d_iwins);
	hipFree(d_pcounts);
	bulk_free(h_pcounts);
}  // count_points_dev

template<class T>
void MafiaSolver<T>::free_dev_resources() {
	hipFree(d_ps);
	hipFree(d_bmps);
}  // free_dev_resources

// explicit instantiations
template class MafiaSolver<float>;
template class MafiaSolver<double>;
  
#endif
